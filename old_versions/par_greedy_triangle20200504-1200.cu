/*
*  par_greedy_triangle.cu
*  Parallel GPU version
*  Author: Randy Shoemaker
*  Some code borrowed from the serial version.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <stdbool.h>
#include <sys/time.h>

// Timer marcros written by Professor Mike Lam of JMU
#define START_TIMER(NAME) gettimeofday(&tv, NULL); \
	double NAME ## _time = tv.tv_sec+(tv.tv_usec/1000000.0);
#define STOP_TIMER(NAME) gettimeofday(&tv, NULL); \
	NAME ## _time = tv.tv_sec+(tv.tv_usec/1000000.0) - (NAME ## _time);
#define GET_TIMER(NAME) (NAME##_time)


  //                                     //
 //             Structs                 //
//	                                   //
/*
 * Structs from the serial version, written by Eliza, Randy, & Alex
*/

/*
 * Represents a point in the plane using cartesian coordinates.
 */
typedef struct {
	double x;
	double y;
} point_t;

/*
 * Represents a line segment between two points.
 */
typedef struct {
	point_t *p;
	point_t *q;
	double len;
} line_t;

  //                                     //
 //          Wrapper Functions          //
//	                                   //
/*
 * Wrappers from the serial version, written by Eliza, Randy, & Alex
*/

/*
 * Prints a point.
 */
void print_point(point_t* point) {
	printf(" (%f , %f) ", point->x, point->y);
}

/*
 * Prints a line.
 */
void print_line(line_t* line) {

	print_point(line->p);
	print_point(line->q);
	printf("\n");
}

/* 
 * Computes the Euclidian distance between two points.
 */
double distance(point_t* p, point_t* q) {
	double delta_x = p->x - q->x;
	double delta_y = p->y - q->y;
	return sqrt(delta_x * delta_x + delta_y * delta_y);
}

/* 
 * Compares two lines. Used for sorting lines with qsort.
 */
int compare(const void* a, const void* b) {
	double result = ((line_t*) a)->len - ((line_t*) b)->len;
	// Make sure that cmp returns an int.
	if (result < 0) {
		return -1;
	}
	else if (result > 0) {
		return 1;
	}
	else {
		return 0;
	}
}

/*
 * Copies 'size' values of 'from' to array 'to'
 */
void copy_array(line_t from[], line_t to[], int size) {
	for (int i = 0; i < size; i++) {
		to[i] = from[i];
	}
}

/*
 * Wrapper function for calloc. It checks for errors as well.
 */
void* allocate(size_t size) {
	void* address = malloc(size);
	if (!address) 	{
		fprintf(stderr, "Cannot malloc, out of memory\n");
		exit(EXIT_FAILURE);
	}
	
	memset(address, 0, size);
	return address;
}

/*
 * Prints a list of lines where each line is stored as an
 * array of four doubles. Author: Randy
 */
void print_lines(double *l, int num_lines) {
	for (int i = 0; i< num_lines; i++) {
		printf("(%lf, %lf) (%lf, %lf)\n", l[4*i],   l[4*i+1],
										  l[4*i+2], l[4*i+3]);
	}
}

  //                         //
 //       DEVICE CODE       //
//                         //
/*
 * Device code written by Randy
*/

__device__ void d_print_lines(double *l, int num_lines) {
	for (int i = 0; i< num_lines; i++) {
		printf("(%lf, %lf) (%lf, %lf)\n", l[4*i],   l[4*i+1],
										  l[4*i+2], l[4*i+3]);
	}
}

// Device function for determining the equality of two points
// p and q. The points are arrays of two ints.
__device__ bool d_is_equal(double *p, double *q) {
	return (p[0] == q[0]) && (p[1] == q[1]);
}

// Device function for determining if the lines I,J share any
// endpoints. The lines are arrays of four ints.
__device__ bool d_share_endpoint(double *I, double *J) {
	// See if any of the end points of I and J are equal.
	// The first two ints in a line are its x-coordinate
	// and the second two ints in a line are its y-coord-
	// -inate. So a line (in this function) is described
	// by four ints.
	return    d_is_equal(&I[0], &J[0]) || d_is_equal(&I[0], &J[2])
		   || d_is_equal(&I[2], &J[0]) || d_is_equal(&I[2], &J[2]);
}

// Helper sign function because nvcc doesnt let me use sign
__device__ int d_sign(double x) { 
	int t = x<0 ? -1 : 0;
	return x > 0 ? 1 : t;
}

// The function for computing orient. Returns 0 if p,q,r are
// colinear, 1 if the traversal from p to q to r is clock-
// -wise, -1 if the traversal from p to q to r is CCW.
__device__ int d_orient(double *p, double *q, double *r) {
	double o = (q[1]-p[1])*(r[0]-q[0]) - (q[0]-p[0])*(r[1]-q[1]);
	// Use sign(float x) to avoid conditional branching
	return d_sign(o);
}

// Device code for finding the max of two doubles 
__device__ double d_max(double a, double b) {
	return (a > b) ? a : b;
}

// Device code for finding the min of two doubles
__device__ double d_min(double a, double b) {
	return (a < b) ? a : b;
}

// The function for computing lies_on. Returns true iff
// the point q lies on the line segment formed by p and r.
__device__ bool d_lies_on(double *p, double *q, double *r) {
	return (q[0] <= d_max(p[0], r[0]) &&
			q[0] >= d_min(p[0], r[0]) &&
			q[1] <= d_max(p[1], r[1]) &&
			q[1] >= d_min(p[1], r[1])
		   );
}

// Checks if the lines I,J intersect
__device__ bool d_intersects(double *I, double *J) {
	// Get the orientation of I_p, I,q, J_p
	int o1 = d_orient(&I[0], &I[2], &J[0]);
	// Get the orientation of I_p, I_q, J_q
	int o2 = d_orient(&I[0], &I[2], &J[2]);
	// Get the orientation of J_p, J_q, I_p
	int o3 = d_orient(&J[0], &J[2], &I[0]);
	// Get the orientation of J_p, J_q, I_q
	int o4 = d_orient(&J[0], &J[2], &I[2]);
	return    (o1 != o2 && o3 != o4)   
		   || (o1 == 0 && d_lies_on(&I[0], &J[0], &I[2]))
		   || (o2 == 0 && d_lies_on(&I[0], &J[2], &I[2]))
		   || (o3 == 0 && d_lies_on(&J[0], &I[0], &J[2]))
		   || (o4 == 0 && d_lies_on(&J[0], &I[2], &J[2]));
}

__device__ bool d_conflicts(double *I, double *J) {
	return !d_share_endpoint(I, J) && d_intersects(I, J);
}

__global__ void triangulate(double *L, int *num_l, int *num_t) {
	int k = threadIdx.x; // A thread's index
	int num_treads = *num_t; // Number of threads
	int lpt = *num_l/num_treads + 1; // Find ceiling of # lines per thread
	int lb_k = k*lpt;     // Lowerbound index of lines for thread k
	int ub_k = (k+1)*lpt; // Upperbound index of lines for thread k

	__shared__ int smallest_idx; // Index of smallest line
	smallest_idx = 0;
	while (smallest_idx < *num_l) {
		for (int j = lb_k; j < ub_k; j++) {
			// Make sure we dont index past the end of the array
			if (j < *num_l) {
				// Run intersection test
				if (d_conflicts(&L[4*smallest_idx], &L[4*j])) {
					// There is an intersection so set line j to be
					// the 'empty line'
					L[4*j]     = 0;
					L[4*j + 1] = 0;
					L[4*j + 2] = 0;
					L[4*j + 3] = 0;
				}
			} 
		}
		// Sync all threads
		__syncthreads();
		// Have one thread increment smallest_idx
		if (threadIdx.x == 0) {
			smallest_idx++;
			while(smallest_idx < *num_l
			   && L[4*smallest_idx]     == 0 
			   && L[4*smallest_idx + 1] == 0
			   && L[4*smallest_idx + 2] == 0
			   && L[4*smallest_idx + 3] == 0) {
				// Skip over all the 'empty' lines
				smallest_idx++;
			}
		}
		// Sync all threads
		__syncthreads();
	}
	//TODO remove all empty lines before moving data back to host.
}

  //                         //
 //          MAIN           //
//                         //

int main(int argc, char *argv[]) {
	
	// Make sure we get the expected input.
	if (argc != 3) {
		printf("Usage %s <filename>, test<>pts<>, num_treads\n", argv[0]);
		exit(EXIT_FAILURE);
	}
	int num_threads = strtol(argv[2], NULL, 10);
	printf("num_threads = %d\n", num_threads);

	  //                         //
	 //  Read points from file  //
	//                         // 
	
	// Open the input file for reading. 
	char *fn = argv[1];
	FILE* fin = fopen(fn, "r");
	if (!fin) {
		fprintf(stderr, "ERROR: Could not open %s\n", fn);
		exit(EXIT_FAILURE);
	}

	long num_points;
	fscanf(fin, "%ld\n", &num_points);
	point_t* points = (point_t*) allocate(num_points * sizeof(point_t));
	
	// Read in and store the point s.
	double x, y;   // The Cartesian coordinates of a point.
	long i = 0;    // Index for storing points.

	while (fscanf(fin, "%lf %lf\n", &x, &y) == 2) {
		// Put the values in a point struct and store.
		point_t *p = (point_t*) allocate(sizeof(point_t));
		p->x = x;
		p->y = y;
		
		// Make sure input file didn't make num_points too small.
		if (i >= num_points) {
			fprintf(stderr, "%s", "ERROR: the number of lines exceeds expectation\n");
			exit(EXIT_FAILURE);
		}
		
		points[i] = *p;
		i++;
		free(p);
	}
	fclose(fin);
	
	  //                      //
	 //  Generate all lines  //
	//                      //
	
	// utility struct for timing calls
	struct timeval tv;
	START_TIMER(generate)
	// Make all possible line segments between the points
	// and compute the length of each line.
	int num_lines = ((num_points)*(num_points-1))/2;
	line_t* lines = (line_t*) allocate(num_lines * sizeof(line_t));
	
	long index = 0;
	for (int i = 0; i < num_points; i++) {
		for (int j = i+1; j < num_points; j++) {
			double length = distance(&points[i], &points[j]);
			line_t* l = (line_t*) allocate(sizeof(line_t));
			// set the values of the line and store it.
			l->p =         &points[i];
			l->q =         &points[j];
			l->len =       length;
			lines[index] = *l;
			index++;
			free(l);
		}
	}
	STOP_TIMER(generate)
	
	  //                                      //
	 //  Sort the lines from small to large  //
	//                                      //
	
	START_TIMER(sort)
	qsort(lines, num_lines, sizeof(line_t), compare);
	STOP_TIMER(sort)
		
	  //                                   //
	 //  Greedily build the tringulation  //
	//	                                 //
	
	START_TIMER(triangulate)
	// The triangulation will be stored as an array of 4 doubles. The doubles
	// give the coordinates of the end points. The length is left out so the
	// ammount of data is decreased by 20%.
	int size = sizeof(double);
	double *segments = (double*) allocate(num_lines*size*4);
	//Transform the lines into arrays of doubles
	for (int i = 0; i < num_lines; i++) {
		point_t p = *(lines[i].p);
		point_t q = *(lines[i].q);
		segments[4*i]   = p.x;
		segments[4*i+1] = p.y;
		segments[4*i+2] = q.x;
		segments[4*i+3] = q.y;
	}

	// Device copies
	double *d_lines;
	int *d_num_lines, *d_num_threads;

	// Allocate space on the device
	hipMalloc((void **)&d_lines, num_lines*4*size);
	hipMalloc((void **)&d_num_lines, sizeof(int));
	hipMalloc((void **)&d_num_threads, sizeof(int));

	// Copy the lines into device memory
	hipMemcpy(d_lines, segments, num_lines*4*size, hipMemcpyHostToDevice);
	hipMemcpy(d_num_lines, &num_lines, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_num_threads, &num_threads, sizeof(int), hipMemcpyHostToDevice);

	// Call the parallel triangulation function 
	triangulate<<<1, num_threads>>>(d_lines, d_num_lines, d_num_threads);

	// Read in the triangulation from the device
	hipMemcpy(segments, d_lines, num_lines*4*size, hipMemcpyDeviceToHost);
	
	STOP_TIMER(triangulate)
	
	  //                                     //
	 //  Triangulation Done, Display Stats  //
	//	                                   //
	
	// These stats are only for the portions of the code specific to the three
	// phases of building the greedy triangulation. Generate all lines, sort the
	// lines in non-decreasing order, and greedily adding line segments to the
	// triangulation.
	printf("Gent: %.4f  Sort: %.4f  Tria: %.4f\n",
			GET_TIMER(generate), GET_TIMER(sort), GET_TIMER(triangulate));

	// Copy the non-empty triangluation lines over to triang array
	int tlines = 0;
	double* triang   = (double*) allocate(num_lines*size*4);
	for (int i = 0; i < num_lines; i++) {
		if (   !(segments[4*i]   == 0) || !(segments[4*i+1] == 0) 
			|| !(segments[4*i+2] == 0) || !(segments[4*i+3] == 0)) {
			//printf("Line %d is non-empty\n", i);
			triang[4*tlines]     = segments[4*i];
			triang[4*tlines + 1] = segments[4*i + 1];
			triang[4*tlines + 2] = segments[4*i + 2];
			triang[4*tlines + 3] = segments[4*i + 3];
			tlines++; // keep track of number of good lines
		}
	}
	// printf("Lines stored in triang before writting file:\n");
	// print_lines(triang, tlines);
	
	// Store the triangulation in a file. 
	FILE* write_file = fopen("triangle_result.txt", "w");
	if (!write_file) {
		fprintf(stderr, "ERROR: Could not open %s\n", "triangle_result.txt");
		exit(EXIT_FAILURE);
	}
	
	// The first line of the file specifies the number of lines in the file.
	fprintf(write_file, "%ld\n", tlines);

	for (int i = 0; i < tlines; i++) {
		// Write the non-empty lines to the file
		fprintf(write_file, "(%lf, %lf) (%lf, %lf)\n", triang[4*i], 
		                                           	   triang[4*i+1], 
		                                               triang[4*i+2], 
		                                               triang[4*i+3]);
	}
	
	fclose(write_file);
	
	// Clean up and exit
	free(triang);
	free(points);
	free(lines);
	return (EXIT_SUCCESS);
}

